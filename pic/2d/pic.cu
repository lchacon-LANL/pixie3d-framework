#include "hip/hip_runtime.h"
#include <unistd.h>
#include <assert.h>
#include <stdio.h>
//#include "cudpp/cudpp.h"
#include "pic_kernel.cuh"
#include "pic.h"
#include "rand.h"

float_p *h_charge;
float_p *h_mass;
int   h_Nsp;
int tot_Np;
float_p h_LL ;
float_p h_hx;
float_p h_ihx;
float_p h_xx0;
float_p h_xf0;
const float_p zero = 0.;


//const float_p h_etola = _fconst(0.015625); //22
//const float_p h_etola = _fconst(0.03125); //22
//const float_p h_etola = _fconst(0.0625); 
//const float_p h_etola = _fconst(0.0078125); 
const float_p h_etola = _fconst(0.0001); 
//const float_p h_etola = _fconst(0.0009765625); 
//const float_p h_etolr = _fconst(0.03); 
//const float_p h_etolr = _fconst(0.03125); 
const float_p h_etolr = _fconst(0.0625); 
//const float_p h_etolr = _fconst(0.015625); 
//const float_p h_etolr = _fconst(0.0078125); 

//const float_p h_etolr = _fconst(0.01); 

const float_p h_eps   =  1e-300; //1e-20; //1e-8; //_fconst(0.0000000001);
const float_p h_PI = _fconst(3.14159265358979323846);

//pointers in gpu global memory
float_p *d_mv,*d_v2; 
float_p *d_E;     //electric field
float_p *d_rho;   //charge density
float_p *d_j;    //current density
float_p *d_xx;    //spatial grid 

int   *d_acNpcls; //number of particles of species (accumulated)
float_p *d_q_m;   //q/m of species
float_p *d_q;     //q of species
float_p *d_m;     //m of species
hipEvent_t     start, stop;

extern "C" {

// void h_move_pcles_acc_cn(Particles pcles, int isp, float_p *E,float_p dt,float_p *j)
// {

//}


  void checkCUDAError(const char *msg)
  {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
      {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
		hipGetErrorString( err) );
        exit(EXIT_FAILURE);
      }                         
  }
  /* end checkCUDAError() */

  void device_start_timing()
  {
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    hipEventRecord( start, 0 );    
  }
  /* end device_start_timing() */

  void device_stop_timing()
  {
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float   elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
    printf( "PIC GPU Time:  %3.1f ms\n", elapsedTime );

    hipEventDestroy( start ) ;
    hipEventDestroy( stop  ) ;
  }
  /* end device_stop_timing() */

void getNBT_re(int n, int &blocks, int &threads)
{        
  if(n==2){
    blocks  = 1;
    threads = 2;
  }else{
    threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
    blocks  = (n + (threads * 2 - 1)) / (threads * 2);
    blocks  = MIN(maxBlocks, blocks);
  }
}
/* end getNBT_re() */

  int h_locate(float_p xp)
  {
#if   _SPORDER == 1
    return      floor((xp-h_xx0)*h_ihx);
#elif _SPORDER == 2
    return       ceil((xp-h_xf0)*h_ihx);
#else 
    printf("Spline order (%d) unavailable in h_locate(),\nEXIT.\n",_SPORDER);
    exit(1);
#endif
  }

//particle B.C.
float_p h_par_bc_pcles(float_p xp)
{
#if   _SPORDER == 1  
  return fmod((xp-h_xx0+h_LL),h_LL)+h_xx0;  
#elif _SPORDER == 2
  xp = fmod((xp-h_xf0+h_LL),h_LL)+h_xf0;  //[xf0,xf0+LL)
  int is_xf0 = (xp==h_xf0);
  return xp + (float_p)(is_xf0)*h_LL; //(xf0,xf0+LL] 
#endif

}
/* end h_par_bc_pcles() */

  void alloc_particles(Particles &pcles,int memSize)
  {
    hipMalloc((void **)&pcles.d_x_n  , memSize);
    hipMalloc((void **)&pcles.d_x_np , memSize);
    hipMalloc((void **)&pcles.d_v_n  , memSize);
    hipMalloc((void **)&pcles.d_v_np , memSize);
    hipMalloc((void **)&pcles.d_ci_n , memSize);
    hipMalloc((void **)&pcles.d_ci_np, memSize);    
  }

  void free_particles(Particles &pcles)
  {
    hipFree(pcles.d_x_n  );
    hipFree(pcles.d_x_np );
    hipFree(pcles.d_v_n  );
    hipFree(pcles.d_v_np );
    hipFree(pcles.d_ci_n );
    hipFree(pcles.d_ci_np);
  }

  void device_finalize()
  {    
    free(h_charge);
    free(h_mass);
#ifdef _USE_TEX
    hipUnbindTexture(d_xxTex);
    hipUnbindTexture(d_ETex);
#endif
    hipFree(d_j    );
    hipFree(d_E    );
    hipFree(d_rho  );
    hipFree(d_xx   );

    //new 
    free_particles(pcles);

    hipFree(d_acNpcls);
    hipFree(d_q_m  );
    hipFree(d_q    );
    hipFree(d_m    );

    hipFree(d_mv);
    checkCUDAError("Free device pointers");
    
  }
  /* end device_finalize() */

  void  device_print_msg()
  {
    cudaPrintfInit();
    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = GRID_SIZE;
      tBlockSize = BLOCK_SIZE;
    }

    device_greetings<<<tGridSize,tBlockSize>>>(d_xx,pcles,0.1);
    checkCUDAError("kernel invocation");
    cudaPrintfDisplay();
    cudaPrintfEnd();   
  }
  /*end device_print_msg() */



  //setup grid and some parameters
  void device_setup_pic(int nx,float_p xx[],bool &explicit_pic,bool &quiet_start, float_p q_m[],float_p v0[],float_p vth[], int npcles[],int &n_sp)
  //  void device_setup_pic(int nxin,float_p xin[],bool explicit,bool quiet_start, float_p q_m[], int npcls[],int nsp)
  {
    h_Nsp = n_sp;

    //printf("#explicit=%s,quiet_start=%s\n",explicit_pic?"t":"f",quiet_start?"t":"f");
    // for(int i=0;i<n_sp;i++){
    //   printf("#spes:%d, q/m=%f, npcles=%d, v0=%f, vth=%f\n\n\n",i,q_m[i],npcles[i],v0[i],vth[i]);
    // }

    
    //copy xx to device
    assert(nx%2==0); //ensure power of 2 (periodic B.C.)
    int memSize = sizeof(float_p)*(nx+2);
    hipMalloc((void **)&d_xx, memSize);        
    hipMemcpy(d_xx,xx,memSize,hipMemcpyHostToDevice);      
    checkCUDAError("copy grid (xx) memory");
#ifdef _USE_TEX
    //texture for grid 
    hipBindTexture( NULL, d_xxTex, d_xx, memSize );
    checkCUDAError("bind grid (xx) texture memory");
#endif

    //set grid parameters
    h_LL = xx[nx] - xx[0];
    h_hx  = xx[1]-xx[0];
    h_ihx = _fconst(1.0)/h_hx;
    h_xf0= _fconst(0.5)*(xx[1]+xx[0]);
    h_xx0= xx[0];
    //printf("#h=%26.16e, ih=%26.16e, xf0=%26.16e, LL=%26.16e\n",h_hx,h_ihx,h_xf0,h_LL);
    //exit(1);
    //single-precision const.
    float h_etolr_sp = float (h_etolr);
    float h_etola_sp = float (h_etola);
    float h_hx_sp = float (h_hx);
    float h_ihx_sp = float (h_ihx);
    hipMemcpyToSymbol(HIP_SYMBOL("etolr_sp"),&h_etolr_sp, sizeof(float),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("etola_sp"),&h_etola_sp, sizeof(float),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("hx_sp"),&h_hx_sp, sizeof(float),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("ihx_sp"),&h_ihx_sp, sizeof(float),
		       0,hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL("etolr"),&h_etolr, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("etola"),&h_etola, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("eps"),&h_eps, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("PI"),&h_PI, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("LL"),&h_LL, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("hx"),&h_hx, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("ihx"),&h_ihx, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("xf0"),&h_xf0, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("xx0"),&h_xx0, sizeof(float_p),
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("Nx"),&nx, sizeof(int),
		       0,hipMemcpyHostToDevice);
    checkCUDAError("set constant memory");

    //Find PIC physical quantities
    float_p omega, n0;
    h_charge=(float_p *)malloc(n_sp*sizeof(float_p));
    h_mass  =(float_p *)malloc(n_sp*sizeof(float_p));
    
    for(int i=0; i<n_sp; i++){
      //omega = sqrt(abs(q_m[i]));
      //n0        = float_p(npcles[i])/h_LL;
      h_charge[i] = (q_m[i]>0?_fconst(1.0):-_fconst(1.0))*h_LL/npcles[i];
      h_mass[i]   = h_charge[i]/q_m[i];
      printf("sp%d: charge=%26.16e, mass=%26.16e \n",i, h_charge[i],h_mass[i]);
    }
    //exit(1);
    //single-precision constants
    float *q_m_sp = (float *) malloc(n_sp*sizeof(float));
    for(int i=0; i<n_sp; i++){
      q_m_sp[i] =  float(q_m[i]);
    }
    printf("%f %f\n",q_m_sp[0],q_m_sp[1]);
    
    hipMemcpyToSymbol(HIP_SYMBOL("dc_q_m_sp"),q_m_sp, sizeof(float)*n_sp,
		       0,hipMemcpyHostToDevice);
    free(q_m_sp);

    //q/m of particle species
    hipMemcpyToSymbol(HIP_SYMBOL("dc_q_m"),q_m, sizeof(float_p)*n_sp,
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("dc_q"),h_charge, sizeof(float_p)*n_sp,
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("dc_m"),h_mass, sizeof(float_p)*n_sp,
		       0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("dc_vth"),vth, sizeof(float_p)*n_sp,
    		       0,hipMemcpyHostToDevice);


     memSize = sizeof(float_p)*n_sp;
     hipMalloc((void**)&d_q_m, memSize);
     hipMemcpy(d_q_m,q_m   ,memSize,hipMemcpyHostToDevice);      
     //q
     hipMalloc((void**)&d_q, memSize);
     hipMemcpy(d_q,h_charge,memSize,hipMemcpyHostToDevice);      
     //m
     hipMalloc((void**)&d_m, memSize);
     hipMemcpy(d_m,h_mass  ,memSize,hipMemcpyHostToDevice);      


    //load particles
     device_load_pcles(v0,vth,npcles,n_sp,xx,nx);


    
  }
  /* end device_setup_pic() */


  void device_load_pcles(float_p v0[],float_p vth[],int numParticles[], int n_sp, float_p xx[], int nx) 
   {
     //set particle numbers
     tot_Np = 0;
     int *acNpcles=(int *)malloc((n_sp+1)*sizeof(int));
     acNpcles[0] = 0;
     for(int i=0; i<n_sp; i++){
       tot_Np += numParticles[i];
       acNpcles[i+1] = tot_Np; //accumulated number
     }

     //number of particles of species
     int memSize = sizeof(int)*(n_sp+1);
     hipMalloc((void**)&d_acNpcls, memSize);
     hipMemcpy(d_acNpcls,acNpcles,memSize,hipMemcpyHostToDevice); 
     hipMemcpyToSymbol(HIP_SYMBOL("dc_acNpcls"),acNpcles, memSize,
			0,hipMemcpyHostToDevice);

     
     hipMemcpyToSymbol(HIP_SYMBOL("dc_Npcls"),numParticles, sizeof(int)*n_sp,
			0,hipMemcpyHostToDevice);
     
     
     hipMemcpyToSymbol(HIP_SYMBOL("totNp"),&tot_Np, sizeof(int),
			0,hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL("Nsp"),&n_sp, sizeof(int),
			0,hipMemcpyHostToDevice);

     checkCUDAError("set constant memory");
     
     //allocate GPU memory             
     //new 
     memSize = sizeof(float_p)*tot_Np*DIM_PIC; 
     int memSize_i = sizeof(int)*tot_Np*DIM_PIC; //cin, cinp
     alloc_particles(pcles,memSize);
     checkCUDAError("Allocate Particles on device");

     //allocate host memory
     float_p *h_x=(float_p *)malloc(memSize);
     float_p *h_v=(float_p *)malloc(memSize);
     int *h_i=(int *)malloc(memSize_i);


     //FILE *fp = fopen("xv.dat","w");
     int ip ,ip_x,ip_v, ci;     
     float_p v_avg = 0;
     float_p dx,xp_off,xp_off0;
     int seed, seedn;

     if(tot_Np==2){
       //2 particles
       h_x[0] = 0.6; //x
       h_x[0] = 0.0; //v
       h_v[1] = 0.8; //x
       h_v[1] = 0.0; //v       
       h_i[0] = h_locate(h_x[0]); //ci_n, ci_np
       h_i[1] = h_locate(h_x[1]);
     }else{
       double rx;
       double rx0=0.0; //1.0;
       ip = 0;
       for(int is=0; is<n_sp; is++){
       seed  = 1;
       seedn = 1;

#if _SPORDER%2==0
	  ci = 1;
#else
	  ci = 0;
#endif
	  int Ni=numParticles[is]/nx;
	  dx = h_LL/numParticles[is];
	  xp_off0 = xp_off = 0.;

	  for(int i=0; i<numParticles[is]; i++){

#ifndef _OFFSET
	   printf("use _OFFSET. exit.\n");
	   exit(1);
#else
	   	   
	   //if(((i+1) & 1) == 0){
	     xp_off+= dx;
	     //}

	   if(xp_off>h_hx){ //careful
	     xp_off -= h_hx;
	     ci++;
	   //B.C. (peroidic, assuming nxx power of 2)
# if _SPORDER%2==0
	     ci = ((ci-1)&(nx-1)) + 1; //from 1 to nx
# else
	     ci = (ci)&(nx-1); //from 0 to nx-1
# endif
	   }
	   /*
# if _SPORDER%2==0
 	     ci = ((ci-1)&(nx-1)) + 1; //from 1 to nx
# else
  	     ci = (ci)&(nx-1); //from 0 to nx-1
# endif
	     //LCG_rand(rx,seed);
	     //Randoms(rx,seed);	     
	     //lcg_rand(rx0,seed);
	     rx0 = float_p(rand())/float_p(RAND_MAX+1);
	     xp_off = h_hx*float_p(rx0);
	   */

 	   h_i[ip] = ci; 
 	   h_x[ip] = xp_off;	   	   
#endif
	   //set velocity
	   //if((i & 1) == 0){
	     randn(rx,seedn);
	     h_v[ip] = v0[is] + vth[is]*float_p(rx); 
	   // }else{ //quiet
	   //   h_v[ip] = h_v[ip-1];
	   // }

	    v_avg += h_v[ip]*h_v[ip];

	    //ci = rand()%nx + 1; //1 to nx
	    //if((i & 1) == 1){
	    // if(((ip+1)%Ni) == 0){
	    //    ci++;
	    // }
	   ip++;
	  }
	  //printf("sum_cpu = %e\n",v_avg*0.5*h_mass[is]);   
	  //v_avg = 0;
       } 



       /*
       for(int is=0; is<n_sp; is++){
	 for(int i=0; i<numParticles[is]; i++){
	   h_x[ip] = h_xx0 + 1e-3 + i*h_LL/numParticles[is]; //x (careful)
	   randn(rx);
	   h_v[ip] = v0[is] + vth[is]*float_p(rx); //v
	   //h_v[ip] = float_p(rx); //v
	   //B.C.
	   h_x[ip] = h_par_bc_pcles(h_x[ip]);
	   h_i[ip] = h_locate(h_x[ip]);//ci_n&np
#ifdef _OFFSET
# if _SPORDER%2==0
	   h_i[ip] = ((h_i[ip]-1) & (nx-1)) + 1; //from 1 to nx
	   h_x[ip] = h_x[ip] - xx[h_i[ip]] + 0.5*h_hx;
# else
	   //peroidic, assuming nxx power of 2           
	   h_i[ip] = (h_i[ip] & (nx-1)); //from 0 to nx-1
	   //offset to the nearest grid point
	   h_x[ip]-= xx[h_i[ip]];
# endif
#endif
	   //fprintf(fp,"%d %d %f %f %d\n",i, ip,h_x[ip],h_v[ip],h_i[ip]);
	   // v_avg += h_v[ip]*h_v[ip];
	   ip++;	   
	 }
	 //fprintf(fp,"\n\n");
	 // printf("sum_cpu = %e\n",v_avg*0.5*h_mass[is]);
	 // v_avg = 0.;
       }
       */
       //printf("#Initial x&v(tot_Np>2) completed.\n");
       
     }

     //copy the particle x&v to device (only this once)
     //new
     hipMemcpy(pcles.d_x_n  ,h_x,memSize  ,hipMemcpyHostToDevice);
     hipMemcpy(pcles.d_x_np ,h_x,memSize  ,hipMemcpyHostToDevice);
     hipMemcpy(pcles.d_v_n  ,h_v,memSize  ,hipMemcpyHostToDevice);
     hipMemcpy(pcles.d_v_np ,h_v,memSize  ,hipMemcpyHostToDevice);
     hipMemcpy(pcles.d_ci_n ,h_i,memSize_i,hipMemcpyHostToDevice);
     hipMemcpy(pcles.d_ci_np,h_i,memSize_i,hipMemcpyHostToDevice);

     checkCUDAError("copy particle (ci) memory");
     
     //new 
     free(h_x );
     free(h_v );
     free(h_i);

     free(acNpcles);
     //fclose(fp);

     //sleep(3);

   }
   /* end device_load_pcles() */

  void device_allocate_field_memory(int nx)
  {

    size_t memSize = sizeof(float_p)*(nx+2);
    hipMalloc((void **)&d_E  , memSize);        
#ifdef _USE_TEX
    //texture for field
    hipBindTexture( NULL, d_ETex, d_E, memSize );
    checkCUDAError("bind field (E) texture memory");
#endif
    checkCUDAError("allocate field (E) device memory");

    //each species has a set of rho for all blocks (careful)
    //    memSize = sizeof(float_p)*(nx+1)*h_Nsp*GRID_SIZE;
    memSize = sizeof(float_p)*(nx)*h_Nsp*GRID_SIZE;

    hipMalloc((void **)&d_rho, memSize);            
    hipMemset( d_rho, zero, memSize );
    checkCUDAError("allocate charge density (rho) device memory");

    memSize *= 2; //p-m
    hipMalloc((void **)&d_j, memSize);        
    hipMemset( d_j, zero, memSize );
    checkCUDAError("allocate current density (j) device memory");

  }
  /* end device_allocate_field_memory() */

  void device_find_current(float_p j[], int nx)
  {
    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = GRID_SIZE;
      tBlockSize = BLOCK_SIZE;
    }

    for(int isp=0; isp<h_Nsp; isp++){    
      find_current2<<<tGridSize,tBlockSize>>>(pcles, isp, d_j);
    }

    //merge d_j
    merge_d_rho<<<1,nx>>>(d_j);
    hipMemcpy(j, d_j, sizeof(float_p)*(nx), hipMemcpyDeviceToHost);
    checkCUDAError("find_current");

    //B.C.
    j[nx]   = j[0];
    j[nx+1] = j[1];


    // for(int i=0; i<nx+2; i++){
    //   printf("%d %f\n",i,j[i]);
    // }
    // exit(0);

  }
  /* end device_find_current() */

  void device_find_rho(float_p rho[], int nx)
  {
    //    printf("#in device_find_rho\n");
    cudaPrintfInit();
    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = GRID_SIZE;
      tBlockSize = BLOCK_SIZE;
    }

    for(int isp=0; isp<h_Nsp; isp++){    
      find_rho<<<tGridSize,tBlockSize>>>(pcles, isp, d_rho);
    }

    //merge d_rho
    merge_d_rho<<<1,nx>>>(d_rho);
    
    hipMemcpy(rho, d_rho, sizeof(float_p)*(nx), hipMemcpyDeviceToHost);
    
    //B.C.
    rho[nx]  = rho[0];
    rho[nx+1]= rho[1];
    

    cudaPrintfDisplay();
    cudaPrintfEnd();  

     // for(int i=0;i<nx+2;i++){
     //   printf("%d %f\n",i,rho[i]);
     // }
     // printf("\n\n");
     // exit(1);
  }
  /* end device_find_rho() */

  //perturb the particle positions
  void device_perturb_pcles(short nh1,float_p eps,float_p dt,float_p rho[],int nx)
  {
    //cudaPrintfInit();
    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = 1; //GRID_SIZE;
      tBlockSize = 32; //BLOCK_SIZE;
    }
    //eps = 0.;
    //perturb_pcles<<<tGridSize,tBlockSize>>>( nh1, eps, pcles, d_acNpcls, d_xx, dt, d_q, d_rho);
    size_t memSize = sizeof(float_p)*(nx)*h_Nsp*GRID_SIZE;

    for(int isp=0; isp<h_Nsp; isp++){
      perturb_pcles2<<<tGridSize,tBlockSize>>>( nh1, eps, pcles, isp, d_xx, dt, d_rho);
    }

    //merge d_rho
    merge_d_rho<<<1,nx>>>(d_rho);

    // cudaPrintfDisplay();
    // cudaPrintfEnd();  

    hipMemcpy(rho, d_rho, sizeof(float_p)*(nx), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCUDAError("perturb particles");
    //B.C.
    rho[nx  ] = rho[0];
    rho[nx+1] = rho[1];



    //       for(int i=0; i<nx+2; i++){
    //   printf("%d %e\n",i,rho[i]);
    // }
    //exit(0);

  }
   /* end device__perturb_pcles() */

void device_set_field(float_p E[], int nx, int itime) 
  {
    //printf("#set device memory for E ...\n");
    int memSize = sizeof(float_p)*(nx+2);
    //copy E to device
    hipMemcpy(d_E  ,E  ,memSize,hipMemcpyHostToDevice);     
    checkCUDAError("copy E to d_E");
    
    //hipMemcpy(d_rho,rho,memSize,hipMemcpyHostToDevice);      
    //hipMemcpy(rho, d_rho, memSize, hipMemcpyDeviceToHost);
    //reset_mem<<<1,nx+2>>>( d_rho, nxp2, 0. );
				                         
    //    if(itime == 713){
    //for(int i=0;i<nx+2;i++){
    // printf("%d %24.16e\n",i,E[i]);
    //}
    // printf("#setupNonlinearFunction after mover.\n");     
    //printf("\n\n");
    //}
  }
  /* end device_set_field() */

  void device_move_pcles(float_p dt, float_p rho[], int nx)
  {
    //cudaPrintfInit();
    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = GRID_SIZE;
      tBlockSize = BLOCK_SIZE;
    }


    size_t memSize = sizeof(float_p)*(nx)*h_Nsp*GRID_SIZE;

    for(int isp=0; isp<h_Nsp; isp++){
#ifdef _OFFSET
      move_pcles_offset<<<tGridSize,tBlockSize>>>(pcles, isp, d_xx, d_E,dt, d_rho);
#else
      move_pcles2<<<tGridSize,tBlockSize>>>(pcles, isp, d_xx,d_q_m, d_E,dt, d_q, d_rho);
#endif
    }
      //merge d_rho
    merge_d_rho<<<1,nx>>>(d_rho);


    hipMemcpy(rho, d_rho, sizeof(float_p)*(nx), hipMemcpyDeviceToHost);


    //free(part_rho);    
    //B.C.
    rho[nx]  = rho[0];
    rho[nx+1]= rho[1];
    
    // printf("\n\n");

    // for(int i=0;i<nx+2;i++){
    //    printf("%d %f\n",i,rho[i]);
    // }
    //  printf("\n\n");

    // exit(1);
  }
  /* end device_move_pcles() */

  //push velocity backward half timestep 
  void device_init_vp_exp(float_p dt)
  {
    cudaPrintfInit();
    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = GRID_SIZE;
      tBlockSize = BLOCK_SIZE;
    }

    init_vp_exp<<<tGridSize,tBlockSize>>>(pcles, d_acNpcls, d_xx,d_q_m, d_E,dt);
    
    cudaPrintfDisplay();
    cudaPrintfEnd();    
    
  }
  /* end device_init_vp_exp() */

  //find moments of each species
  void device_moments(float_p *moments,float_p *ki_en)
  {
     int numBlocks=16, numThreads=64;
     //getNBT_re(tot_Np, numBlocks, numThreads);  
     

     //for toal momentum
     hipMalloc((void **)&d_mv, sizeof(float_p)*numBlocks*h_Nsp); 

     for(int isp=0; isp<h_Nsp; isp++){
       output_mv<<<numBlocks, numThreads,
    	 numThreads * sizeof(float_p)>>>(pcles.d_v_n,isp,d_mv,numThreads); //careful
     }


     float_p h_part_sum[numBlocks*h_Nsp];
     hipMemcpy(&h_part_sum, d_mv, sizeof(float_p) * numBlocks*h_Nsp,
     		hipMemcpyDeviceToHost);
     float_p final_sum = 0;
     for(int i = 0; i < numBlocks*h_Nsp; i++) {
       final_sum += h_part_sum[i];
     }
     moments[1] = final_sum;


    //sum up kineitc energy
     hipMalloc((void **)&d_v2, sizeof(float_p)*numBlocks*h_Nsp); 
     moments[2] = 0;
     for(int isp=0; isp<h_Nsp; isp++){
       output_v2<<<numBlocks, numThreads,
	 numThreads * sizeof(float_p)>>>(pcles.d_v_n,pcles.d_v_n,isp,d_v2,numThreads); //careful
       hipMemcpy(&h_part_sum[isp*numBlocks], d_v2+isp*numBlocks, sizeof(float_p) * numBlocks, hipMemcpyDeviceToHost);
       final_sum = 0;
       for(int i = isp*numBlocks; i < (isp+1)*numBlocks; i++) {
	 final_sum += h_part_sum[i];
       }
       ki_en[isp] = 0.5*h_mass[isp]*final_sum;
       moments[2]+= ki_en[isp];
     }

  }




/*----- implicit functions ------*/

void device_move_pcles_acc_cn(float_p dt, float_p j[], int nx, int itime,float_p dE[],bool use_sp)
  {
    //cudaPrintfInit();

    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = GRID_SIZE;
      tBlockSize = G_WARP*WARP_SIZE; //BLOCK_SIZE;
    }

    if(use_sp){
      for(int isp=0; isp<h_Nsp; isp++){
	move_pcles_acc_cn_sp<<<tGridSize,tBlockSize,G_WARP*nx*sizeof(float)>>>(pcles, isp, d_E,dt, d_j);
      }
    }else{
      for(int isp=0; isp<h_Nsp; isp++){
      //move_pcles_cn<<<tGridSize,tBlockSize>>>(pcles, isp, d_E,dt, d_j);
	move_pcles_acc_cn<<<tGridSize,tBlockSize,G_WARP*nx*sizeof(float_p)>>>(pcles, isp, d_E,dt, d_j);
      //move_pcles_acc_cn2<<<tGridSize,tBlockSize,G_WARP*nx*sizeof(float_p)>>>(pcles, isp, d_E,dt, d_j);
      //cudaPrintfDisplay();
      }
    }

    //cudaPrintfEnd();    
    hipDeviceSynchronize();
    //merge d_j
    /*
    merge_d_rho<<<1,nx>>>(d_j);

    hipMemcpy(j, d_j, sizeof(float_p)*(nx), hipMemcpyDeviceToHost);

    for(int i=0; i<nx; i++){
      j[i] = j[i]/(h_hx*dt);
    }
    */
    size_t memSize = sizeof(float_p)*(nx)*h_Nsp*GRID_SIZE*2;

    float_p *h_j = (float_p *)malloc(memSize);
    hipMemcpy(h_j, d_j, memSize, hipMemcpyDeviceToHost);
    for(int ix=0; ix<nx; ix++){
      j[ix] = _fconst(0.0);
    }
    //merge
    float_p a,b,c;
    for(int ix=0; ix<nx; ix++){
      float_p e0=_fconst(0.0),e1=_fconst(0.0);
      for(int ib=1; ib<GRID_SIZE; ib++){
	h_j[ix] += h_j[ix+ib*nx];
	h_j[ix+GRID_SIZE*nx] +=  h_j[ix+GRID_SIZE*nx+ib*nx];
	h_j[ix+MS] += h_j[ix+MS+ib*nx];
	h_j[ix+MS+GRID_SIZE*nx] +=  h_j[ix+MS+GRID_SIZE*nx+ib*nx];
	//	printf("%d %d %24.16e  %24.16e \n",ix, ib,h_j[ix+ib*nx],h_j[ix+ib*nx+MS]);

	/*
	c = h_j[ix+ib*nx]; 
	if(_abs(c)>_abs(h_j[ix])){
	  a = c;
	  c = h_j[ix]; 
	}else{
	  a = h_j[ix];
	}
	b = e0 + c;
	h_j[ix] = a+b;
	e0 = b-(h_j[ix]-a);

	c = h_j[ix+GRID_SIZE*nx+ib*nx]; 
	if(_abs(c)>_abs(h_j[ix+GRID_SIZE*nx])){
	  a = c;
	  c = h_j[ix+GRID_SIZE*nx]; 
	}else{	  
	  a = h_j[ix+GRID_SIZE*nx];
	}
	b = e1 + c;
	h_j[ix+GRID_SIZE*nx] = a+b;
	e1 = b-(h_j[ix+GRID_SIZE*nx]-a);
	*/
      }
    }
    for(int ix=0; ix<nx; ix++){
      for(int is=0; is<h_Nsp; is++){
	j[ix] += h_j[ix+is*GRID_SIZE*nx]*h_charge[is];
	//printf("%d %d %24.16e  %24.16e \n",is, ix,h_j[ix+is*GRID_SIZE],j[ix]);
      }
      for(int is=0; is<h_Nsp; is++){
	j[ix] += h_j[MS+ix+is*GRID_SIZE*nx]*h_charge[is];
	//printf("%d %d %24.16e  %24.16e \n",is, ix,h_j[MS+ix+is*GRID_SIZE],j[ix]);
      }

      j[ix] = j[ix]/(h_hx*dt);
    }
    //B.C.
    j[nx]  = j[0];
    j[nx+1]= j[1];
    
    checkCUDAError("move(acc_cn) particles");
    free(h_j);
    //    if(j[20]!=j[20]){
      //if(itime > 713){
    //for(int i=0;i<nx+2;i++){
    // printf("%d %24.16e %24.16e \n",i,dE[i],j[i]);
    //}
    //printf("#%d setupNonlinearFunction after mover.\n",itime);	
    //printf("\n\n");
    // exit(1);
    //    }

  }
  /* end device_move_pcles_acc_cn() */


  void device_update_tnp_pcles()
  {
    int tBlockSize,tGridSize;
    if(tot_Np==2){
      tGridSize  = 1;
      tBlockSize = 2;
    }else{
      tGridSize  = 1; //maxBlocks; //GRID_SIZE;
      tBlockSize = 64; //maxThreads; //BLOCK_SIZE;
    }
    

#ifdef _SORT
    for(int isp=0; isp<h_Nsp; isp++){
      sortv_particles<<<tGridSize,tBlockSize>>>(pcles,isp);
    }
#else
    update_tnp_pcles<<<tGridSize,tBlockSize>>>(pcles);
#endif
  }
  /* end device_update_tnp_pcles() */
}
